﻿
#include "hip/hip_runtime.h"
#include "cuda_device_runtime_api.h"

#include "include/cu_assert.h"
#include "include/config.h"
#include "include/blackhole.cuh"
#include "include/bmp.h"
#include "include/jpeg.h"

void CreateCUDABlackHole(void)
{
	hipSetDevice(0);

	unsigned char* host_bitmap = null;
	unsigned char* dev_image;

#pragma region Time measurement 
	hipEvent_t startTime, stopTime;

	cudaAssert(hipEventCreate(&startTime));
	cudaAssert(hipEventCreate(&stopTime));
	cudaAssert(hipEventRecord(startTime));
#pragma endregion

	hipHostAlloc((void**)&host_bitmap, DIM_X * DIM_Y * 3 * sizeof(unsigned char), hipHostMallocWriteCombined);
	cudaAssert(hipMalloc((void**)&dev_image, DIM_X * DIM_Y * 3 * sizeof(unsigned char)));
	ProcessRayTracing(dev_image);
	cudaAssert(hipMemcpy(host_bitmap, dev_image, DIM_X * DIM_Y * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));
	cudaAssert(hipFree(dev_image));

#pragma region Time measurement 
	cudaAssert(hipEventRecord(stopTime));
	cudaAssert(hipEventSynchronize(stopTime));

	float time;
	cudaAssert(hipEventElapsedTime(&time, startTime, stopTime));

	printf("time: %.2f ms\n", time);
#pragma endregion

#if !defined(JPEG)
	BMPfile_t saveImage;
	CreateBMP(&saveImage, DIM_X, DIM_Y, host_bitmap);
	SaveBMPtoFile(&saveImage, FILENAME);
#else
	SaveJPEGtoFile(host_bitmap, FILENAME);
#endif // BMP

	cudaAssert(hipHostFree(host_bitmap));
	cudaAssert(hipDeviceReset());

}

int main()
{
	CreateCUDABlackHole();
	return 0;
}

/*
int main()
{
	hipSetDevice(0);

	unsigned char* host_bitmap = null;
	hipHostAlloc((void**)&host_bitmap, DIM_X * DIM_Y * 3 * sizeof(unsigned char), hipHostMallocWriteCombined);

	//DataBlock data;
	//CPUBitmap cpuBitmap(host_bitmap, DIM_X, DIM_Y, &data);

	unsigned char*	dev_image;
	float*			dev_imageFloat;

	Camera_t*		camera;
	BlackHole_t*	blackHole;

	Camera_t*		dev_camera;
	BlackHole_t*	dev_blackHole;
	Star_t*			dev_stars;

	unsigned char*	dev_bitmap;

	float*			dev_gaussKernel;
	float*			dev_gaussKernelSmall;

	hiprandStateXORWOW_t* dev_random;

	hipEvent_t startTime, stopTime;

	camera = CreateCamera();
	blackHole = CreateBlackHole();

	SetBlackHole(blackHole, DISK_IN, DISK_OUT);
	
#ifdef RAYTRACING
	//original settings
	//SetCameraParameters(camera, vector3{0.f, 4.3f, -9.f}, FOV_TANGENT, Vector3Zero(), vector3{0.f, 1.f, 0.f});
	SetCameraParameters(camera, vector3{ 0.0f, -4.3f, -18.f }, FOV_TANGENT, Vector3Zero(), vector3{ 0.f, 1.f, 0.f });
#else
#ifdef JPEG
	SetCameraParameters(camera, vector3{ 0.0f, -1.f, -22.f }, FOV_TANGENT, Vector3Zero(), vector3{ 0.2f, -1.f, 0.f });
#else
	SetCameraParameters(camera, vector3{ 0.0f, -1.f, -20.f }, FOV_TANGENT, Vector3Zero(), vector3{ 0.2f, -1.f, 0.f });
#endif // JPEG
	
#endif // RAYTRACING

#pragma region Time measurement 
	cudaAssert(hipEventCreate(&startTime));

	cudaAssert(hipEventCreate(&stopTime));

	cudaAssert(hipEventRecord(startTime));
	
#pragma endregion

#pragma region Device memory allocation

	cudaAssert(hipMalloc((void**)&dev_image, DIM_X * DIM_Y * 3 * sizeof(unsigned char)));
	//data.dev_image = dev_image;

	cudaAssert(hipMalloc((void**)&dev_bitmap, DIM_X_N(BITMAP_MULTIPIER) * DIM_Y_N(BITMAP_MULTIPIER) * 4 * sizeof(unsigned char)));
	
	cudaAssert(hipMalloc((void**)&dev_imageFloat, DIM_X * DIM_Y * 3 * sizeof(float)));
	
	//data.dev_imageFloat = dev_imageFloat;
	
	cudaAssert(hipMalloc((void**)&dev_blackHole, sizeof(BlackHole_t)));

	cudaAssert(hipMalloc((void**)&dev_camera, sizeof(Camera_t)));
	
	cudaAssert(hipMalloc((void**)&dev_stars, DIM_X_N(BITMAP_MULTIPIER) * DIM_Y_N(BITMAP_MULTIPIER) / STAR_DENSITY * sizeof(Star_t)));
	
	cudaAssert(hipMalloc((void**)&dev_random, DIM_X_N(BITMAP_MULTIPIER) * DIM_Y_N(BITMAP_MULTIPIER) / STAR_DENSITY * sizeof(hiprandStateXORWOW_t)));
	

#if defined(GAUSSIAN_ROW_COL)
	cudaAssert(hipMalloc((void**)&dev_gaussKernel, (2 * GAUSS_SIZE + 1) * sizeof(float)));
	
	cudaAssert(hipMalloc((void**)&dev_gaussKernelSmall, (2 * GAUSS_SIZE_SMALL + 1) * sizeof(float)));
	
#else
	cudaAssert(hipMalloc((void**)&dev_gaussKernel, GAUSS_KERNEL_SIZE * sizeof(float)));
	
#endif
	
#pragma endregion
	// blackbody colors 
	cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(colorMap), colors, 500 * sizeof(vector3)));
	
	// small gauss kernel 
	//hipMemcpyToSymbol(HIP_SYMBOL(aaKernel), aaGauss, 49 * sizeof(float));

	//Gaussian efect
	//dim3 kerngrid(2 * GAUSS_SIZE + 1, 2 * GAUSS_SIZE + 1);
	
#if defined(GAUSSIAN_ROW_COL)
	//dim3 kerngrid(2 * GAUSS_SIZE + 1);
	//dim3 kerngrid(GAUSS_SIZE + 1);
	GenerateGaussianKernel1D << < GAUSS_SIZE + 1, 1 >> > (dev_gaussKernel, (float)((GAUSS_SIZE / 3) * (GAUSS_SIZE / 3))); //289 = 17^2 // 1156 = 34^2
	cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(convolveKernel1D), dev_gaussKernel, (2 * GAUSS_SIZE + 1) * sizeof(float), 0, hipMemcpyDeviceToDevice));
	
	GenerateGaussianKernelSmall1D << < GAUSS_SIZE_SMALL + 1, 1 >> > (dev_gaussKernelSmall, (float)((GAUSS_SIZE_SMALL / 3) * (GAUSS_SIZE_SMALL / 3))); //289 = 17^2
	cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(convolveKernelSmall1D), dev_gaussKernelSmall, (2 * GAUSS_SIZE_SMALL + 1) * sizeof(float), 0, hipMemcpyDeviceToDevice));
#else
	dim3 kerngrid(GAUSS_SIZE + 1, GAUSS_SIZE + 1);
	GenerateGaussianKernel << < kerngrid, 1 >> > (dev_gaussKernel, 289.f); //289 = 17^2
	hipMemcpyToSymbol(HIP_SYMBOL(convolveKernel), dev_gaussKernel, GAUSS_KERNEL_SIZE * sizeof(float), 0, hipMemcpyDeviceToDevice);
#endif
	
	dim3 grid(DIM_X / 16, DIM_Y / 16);
	dim3 threads(16, 16);

	dim3 gridBmp(DIM_X_N(BITMAP_MULTIPIER) / 16, DIM_Y_N(BITMAP_MULTIPIER) / 16);
	// threads identical as 'threads'

	dim3 grid1Axis(DIM_X / 16);
	dim3 threads1Axis(16);

	dim3 gridStar(DIM_X_N(BITMAP_MULTIPIER)* DIM_Y_N(BITMAP_MULTIPIER) / STAR_DENSITY / 32);
	// threads identical as 'threads1Axis'

	cudaAssert(hipMemcpy(dev_blackHole, blackHole, sizeof(BlackHole_t), hipMemcpyHostToDevice));
	cudaAssert(hipMemcpy(dev_camera, camera, sizeof(Camera_t), hipMemcpyHostToDevice));
	
	// prepare very dark blue sky
	//PrepareBackground<< <gridBmp, threads >> >(dev_bitmap);
	cudaAssert(hipMemset(dev_bitmap, 0x00, DIM_X_N(BITMAP_MULTIPIER)* DIM_Y_N(BITMAP_MULTIPIER) * sizeof(float)));
	
	// generate some stars
	PrepareRandom << <gridStar, 32 >> > (dev_random);
	GenerateSky << <gridStar, 32 >> > (dev_random, dev_stars);

	// draw stars using CPU on bitmap
	DrawSky << < gridStar, 32 >> >(dev_bitmap, dev_stars);

	// copy bitmap to texture memory
	hipTextureObject_t textureBitmap = 0;

	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(hipResourceDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = dev_bitmap;
	resDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
	resDesc.res.linear.desc.x = 8; // bits per channel
	resDesc.res.linear.desc.y = 8; // bits per channel
	resDesc.res.linear.desc.z = 8; // bits per channel
	resDesc.res.linear.desc.w = 8; // bits per channel
	resDesc.res.linear.sizeInBytes = DIM_X_N(BITMAP_MULTIPIER) * DIM_Y_N(BITMAP_MULTIPIER) * 4 * sizeof(unsigned char);
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(hipTextureDesc));
	texDesc.readMode = hipReadModeElementType;

	cudaAssert(hipCreateTextureObject(&textureBitmap, &resDesc, &texDesc, null));
	
	// render black hole with accretion disc 
	RayTrace << < grid, threads >> > (dev_imageFloat, textureBitmap, dev_camera, dev_blackHole);

	// add small antialiasing using small gauss kernel
	//Antialiasing << < grid, threads >> > (dev_imageFloat);

	// add gauss effect in postprocessing
#if  defined(GAUSSIAN)
	GaussianConvolve << <grid, threads >> > (dev_imageFloat, dev_image);
#elif defined(GAUSSIAN_ROW_COL)
	float* dev_gaussTmp;
	cudaAssert(hipMalloc((void**)&dev_gaussTmp, DIM_X * DIM_Y * 3 * sizeof(float)));

	GaussianConvolveRowSmall << < grid, threads >> > (dev_imageFloat, dev_gaussTmp);
	GaussianConvolveColSmall << < grid, threads >> > (dev_gaussTmp, dev_imageFloat);
	//FinishRayTracing << <grid, threads >> > (dev_imageFloat, dev_image);

	GaussianConvolveRow << < grid, threads >> > (dev_imageFloat, dev_gaussTmp);
	GaussianConvolveCol << < grid, threads >> > (dev_gaussTmp, dev_imageFloat);
#endif // GAUSSIAN
	FinishRayTracing << <grid, threads >> > (dev_imageFloat, dev_image);

	cudaAssert(hipMemcpy(host_bitmap, dev_image, DIM_X * DIM_Y * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));

	cudaAssert(hipEventRecord(stopTime));
	
	cudaAssert(hipEventSynchronize(stopTime));

	float time;
	cudaAssert(hipEventElapsedTime(&time, startTime, stopTime));
	
	printf("time: %.2f ms\n", time);

#pragma region Cleanup CUDA memory
	cudaAssert(hipFree(dev_bitmap));

	cudaAssert(hipFree(dev_blackHole));
	
	cudaAssert(hipFree(dev_camera));
	
	cudaAssert(hipFree(dev_gaussKernel));

	cudaAssert(hipFree(dev_image));
	
	cudaAssert(hipFree(dev_imageFloat));

	cudaAssert(hipFree(dev_random));

	cudaAssert(hipFree(dev_stars));
	
	cudaAssert(hipDestroyTextureObject(textureBitmap));
	
#pragma endregion

#if !defined(JPEG)
	BMPfile_t saveImage;
	CreateBMP(&saveImage, DIM_X, DIM_Y, host_bitmap);
	SaveBMPtoFile(&saveImage, FILENAME);

	cudaAssert(hipHostFree(host_bitmap));
#else
	SaveJPEGtoFile(host_bitmap, FILENAME);
	cudaAssert(hipHostFree(host_bitmap));
#endif // BMP

	cudaAssert(hipDeviceReset());

}
*/